#include "hip/hip_runtime.h"
// Copyright 2010 Rich Townsend <townsend@astro.wisc.edu>
//
// This file is part of CULSP.
//
// CULSP is free software: you can redistribute it and/or modify it
// under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// CULSP is distributed in the hope that it will be useful, but
// WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
// General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with CULSP. If not, see <http://www.gnu.org/licenses/>.

#ifndef _CULSP_KERNEL_BATCH_
#define _CULSP_KERNEL_BATCH_



__global__ void
__launch_bounds__(BLOCK_SIZE)
dummy_kernel(float *d_t, float *d_X, float *d_P, float df, 
                      int offset, int N_t, int N_f, float minf)
{
  /*
  int id = blockIdx.x*BLOCK_SIZE+threadIdx.x;
 
  int i;

  if ( id >= N_f ) return;

  __shared__ float s_t[BLOCK_SIZE];
  __shared__ float s_X[BLOCK_SIZE];

  // Calculate the frequency

  float f = (id+1)*df + minf;

  // Calculate the various sums

  float XC = 0.f;
  float XS = 0.f;
  float CC = 0.f;
  float CS = 0.f;

  float XC_chunk = 0.f;
  float XS_chunk = 0.f;
  float CC_chunk = 0.f;
  float CS_chunk = 0.f;

  int j;
  */
  return;
}

__global__ void
__launch_bounds__(BLOCK_SIZE)
culsp_kernel_stream(float *d_t, float *d_X, float *d_P, float df, 
                      int offset, int N_t, int N_f, float minf)
{
  int id = blockIdx.x*BLOCK_SIZE+threadIdx.x;
 
  int i;

  if ( id >= N_f ) return;

  __shared__ float s_t[BLOCK_SIZE];
  __shared__ float s_X[BLOCK_SIZE];

  // Calculate the frequency

  float f = (id+1)*df + minf;

  // Calculate the various sums

  float XC = 0.f;
  float XS = 0.f;
  float CC = 0.f;
  float CS = 0.f;

  float XC_chunk = 0.f;
  float XS_chunk = 0.f;
  float CC_chunk = 0.f;
  float CS_chunk = 0.f;

  int j;

  for(j = 0; j < N_t- BLOCK_SIZE; j += BLOCK_SIZE) {

    // Load the chunk into shared memory

    __syncthreads();

    s_t[threadIdx.x] = d_t[offset + j+threadIdx.x];
    s_X[threadIdx.x] = d_X[offset + j+threadIdx.x];

    __syncthreads();

    // Update the sums

    #pragma unroll
    for(int k = 0; k < BLOCK_SIZE; k++) {

      // Range reduction

      float ft = f*s_t[k];
      ft -= rintf(ft);

      float c;
      float s;

      __sincosf(TWOPI*ft, &s, &c);

      XC_chunk += s_X[k]*c;
      XS_chunk += s_X[k]*s;
      CC_chunk += c*c;
      CS_chunk += c*s;

    }

    XC += XC_chunk;
    XS += XS_chunk;
    CC += CC_chunk;
    CS += CS_chunk;

    XC_chunk = 0.f;
    XS_chunk = 0.f;
    CC_chunk = 0.f;
    CS_chunk = 0.f;
    
  }

  // Handle the final chunk

  __syncthreads();

  if(j+threadIdx.x < N_t) {

    s_t[threadIdx.x] = d_t[offset + j+threadIdx.x];
    s_X[threadIdx.x] = d_X[offset + j+threadIdx.x];

  }

  __syncthreads();
    
  for(int k = 0; k < N_t-j; k++) {

    // Range reduction

    float ft = f*s_t[k];
    ft -= rintf(ft);

    float c;
    float s;

    __sincosf(TWOPI*ft, &s, &c);

    XC_chunk += s_X[k]*c;
    XS_chunk += s_X[k]*s;
    CC_chunk += c*c;
    CS_chunk += c*s;

  }

  XC += XC_chunk;
  XS += XS_chunk;
  CC += CC_chunk;
  CS += CS_chunk;

  float SS = (float) N_t - CC;
    
  // Calculate the tau terms

  float ct;
  float st;

  __sincosf(0.5f*atan2(2.f*CS, CC-SS), &st, &ct);

  // Calculate P

  d_P[id] = 
      0.5f*((ct*XC + st*XS)*(ct*XC + st*XS)/
      (ct*ct*CC + 2*ct*st*CS + st*st*SS) + 
      (ct*XS - st*XC)*(ct*XS - st*XC)/
      (ct*ct*SS - 2*ct*st*CS + st*st*CC));

  // Finish

}


__global__ void
__launch_bounds__(BLOCK_SIZE)
culsp_kernel_batch(float *d_t, float *d_X, float *d_P, float df, 
                      int *N_t, int Nlc, int N_f, float minf)
{
  int id = blockIdx.x*BLOCK_SIZE+threadIdx.x;
  int lcno = id % N_f;
  int offset = 0;
  int i;
  for(i=0; i<lcno; i++) offset += N_t[i];
 
  if ( id >= N_f*Nlc || lcno >= Nlc ) return;

  __shared__ float s_t[BLOCK_SIZE];
  __shared__ float s_X[BLOCK_SIZE];

  // Calculate the frequency

  float f = (id+1)*df + minf;

  // Calculate the various sums

  float XC = 0.f;
  float XS = 0.f;
  float CC = 0.f;
  float CS = 0.f;

  float XC_chunk = 0.f;
  float XS_chunk = 0.f;
  float CC_chunk = 0.f;
  float CS_chunk = 0.f;

  int j;

  for(j = 0; j < N_t[lcno]-BLOCK_SIZE; j += BLOCK_SIZE) {

    // Load the chunk into shared memory

    __syncthreads();

    s_t[threadIdx.x] = d_t[offset + j+threadIdx.x];
    s_X[threadIdx.x] = d_X[offset + j+threadIdx.x];

    __syncthreads();

    // Update the sums

    #pragma unroll
    for(int k = 0; k < BLOCK_SIZE; k++) {

      // Range reduction

      float ft = f*s_t[k];
      ft -= rintf(ft);

      float c;
      float s;

      __sincosf(TWOPI*ft, &s, &c);

      XC_chunk += s_X[k]*c;
      XS_chunk += s_X[k]*s;
      CC_chunk += c*c;
      CS_chunk += c*s;

    }

    XC += XC_chunk;
    XS += XS_chunk;
    CC += CC_chunk;
    CS += CS_chunk;

    XC_chunk = 0.f;
    XS_chunk = 0.f;
    CC_chunk = 0.f;
    CS_chunk = 0.f;
    
  }

  // Handle the final chunk

  __syncthreads();

  if(j+threadIdx.x < N_t[lcno]) {

    s_t[threadIdx.x] = d_t[offset + j+threadIdx.x];
    s_X[threadIdx.x] = d_X[offset + j+threadIdx.x];

  }

  __syncthreads();
    
  for(int k = 0; k < N_t[lcno]-j; k++) {

    // Range reduction

    float ft = f*s_t[k];
    ft -= rintf(ft);

    float c;
    float s;

    __sincosf(TWOPI*ft, &s, &c);

    XC_chunk += s_X[k]*c;
    XS_chunk += s_X[k]*s;
    CC_chunk += c*c;
    CS_chunk += c*s;

  }

  XC += XC_chunk;
  XS += XS_chunk;
  CC += CC_chunk;
  CS += CS_chunk;

  float SS = (float) N_t[lcno] - CC;
    
  // Calculate the tau terms

  float ct;
  float st;

  __sincosf(0.5f*atan2(2.f*CS, CC-SS), &st, &ct);

  // Calculate P

  d_P[id] = 
      0.5f*((ct*XC + st*XS)*(ct*XC + st*XS)/
	    (ct*ct*CC + 2*ct*st*CS + st*st*SS) + 
	    (ct*XS - st*XC)*(ct*XS - st*XC)/
	    (ct*ct*SS - 2*ct*st*CS + st*st*CC));

  // Finish

}


__global__ void
__launch_bounds__(BLOCK_SIZE)
bootstrap_kernel_batch(float *d_t, float *d_X, float *d_P, float df, 
                  int *N_t, int Nlc, int N_f, float minf, hiprandState *state){

  // Same as culsp kernel, except that we draw an s_X value at random.
  // doing this N times will give you a statistical bootstrap from which
  // false alarm probabilities can be calculated. This gets rid of the Gaussian
  // error assumption, but does NOT relax the assumption that all observations
  // are uncorrelated (violated when you have red noise, etc.)
  int id = blockIdx.x*BLOCK_SIZE+threadIdx.x;
  int lcno = id % N_f;
  int offset = 0;
  int i;
  for (i=0; i < lcno; i++) offset+=N_t[i];
  if ( id >= N_f*Nlc) return;
  
  

  __shared__ float s_t[BLOCK_SIZE];
  __shared__ float s_X[BLOCK_SIZE];

  // Calculate the frequency

  float f = (id+1)*df + minf;

  // Calculate the various sums

  float XC = 0.f;
  float XS = 0.f;
  float CC = 0.f;
  float CS = 0.f;

  float XC_chunk = 0.f;
  float XS_chunk = 0.f;
  float CC_chunk = 0.f;
  float CS_chunk = 0.f;

  int j, jrand;

  for(j = 0; j < N_t[lcno]-BLOCK_SIZE; j += BLOCK_SIZE) {

    // Load the chunk into shared memory

    __syncthreads();

    jrand = (int) (N_t[lcno] * hiprand_uniform(&state[id]) );

    s_t[threadIdx.x] = d_t[offset + j+threadIdx.x];
    s_X[threadIdx.x] = d_X[offset + jrand];

    __syncthreads();

    // Update the sums

    #pragma unroll
    for(int k = 0; k < BLOCK_SIZE; k++) {

      // Range reduction

      float ft = f*s_t[k];
      ft -= rintf(ft);

      float c;
      float s;

      __sincosf(TWOPI*ft, &s, &c);

      XC_chunk += s_X[k]*c;
      XS_chunk += s_X[k]*s;
      CC_chunk += c*c;
      CS_chunk += c*s;

    }

    XC += XC_chunk;
    XS += XS_chunk;
    CC += CC_chunk;
    CS += CS_chunk;

    XC_chunk = 0.f;
    XS_chunk = 0.f;
    CC_chunk = 0.f;
    CS_chunk = 0.f;
    
  }

  // Handle the final chunk

  __syncthreads();

  if(j+threadIdx.x < N_t[lcno]) {
    jrand = (int) (N_t[lcno] * hiprand_uniform(&state[id]));

    s_t[threadIdx.x] = d_t[offset + j+threadIdx.x];
    s_X[threadIdx.x] = d_X[offset + jrand];

  }

  __syncthreads();
    
  for(int k = 0; k < N_t[lcno]-j; k++) {

    // Range reduction

    float ft = f*s_t[k];
    ft -= rintf(ft);

    float c;
    float s;

    __sincosf(TWOPI*ft, &s, &c);

    XC_chunk += s_X[k]*c;
    XS_chunk += s_X[k]*s;
    CC_chunk += c*c;
    CS_chunk += c*s;

  }

  XC += XC_chunk;
  XS += XS_chunk;
  CC += CC_chunk;
  CS += CS_chunk;

  float SS = (float) N_t[lcno] - CC;
    
  // Calculate the tau terms

  float ct;
  float st;

  __sincosf(0.5f*atan2(2.f*CS, CC-SS), &st, &ct);

  // Calculate P

  d_P[id] = 
      0.5f*((ct*XC + st*XS)*(ct*XC + st*XS)/
      (ct*ct*CC + 2*ct*st*CS + st*st*SS) + 
      (ct*XS - st*XC)*(ct*XS - st*XC)/
      (ct*ct*SS - 2*ct*st*CS + st*st*CC));

  // Finish

}



#endif
