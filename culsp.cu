// Copyright 2010 Rich Townsend <townsend@astro.wisc.edu>
//
// This file is part of CULSP.
//
// CULSP is free software: you can redistribute it and/or modify it
// under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// CULSP is distributed in the hope that it will be useful, but
// WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
// General Public License for more details.
//
// You should have received a copy of the GNU General Public License
// along with CULSP. If not, see <http://www.gnu.org/licenses/>.

// Includes

#include <stdlib.h>
#include <string.h>
#include <argtable2.h>

#include "periodogram.h"
#include "culsp_kernel.cu"

// Wrapper macros

#define CUDA_CALL(call) {						\
    hipError_t err = call;						\
    if(err != hipSuccess) {						\
      fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",	\
	      __FILE__, __LINE__, hipGetErrorString(err));		\
      exit(EXIT_FAILURE);						\
    }}

// Forward declarations

void initialize (int, char **, char **, char **, float *, float *, int *);
void initialize_cuda (int, int);
void eval_LS_periodogram (int, int, float, float *, float *, float *);

// Main program

int
main( int argc, char** argv) 
{

  char *filename_in;
  char *filename_out;
  float F_over;
  float F_high;
  int device;

  int N_t;
  int N_f;
 
  float *t;
  float *X;

  float df;
  float *P;

  // Initialize

  initialize(argc, argv, &filename_in, &filename_out, &F_over, &F_high, &device);

  // Read the light curve

  read_light_curve(filename_in, &N_t, &t, &X);

  // Set up the frequency parameters

  set_frequency_params(N_t, t, F_over, F_high, &N_f, &df);

  // Allocate space for the periodogram

  P = (float *) malloc(N_f*sizeof(float));

  // Initialize CUDA

  initialize_cuda(device, N_f);

  // Start the timer

  double time_a = get_time();
		 
  // Evaluate the Lomb-Scargle periodogram
 
  eval_LS_periodogram(N_t, N_f, df, t, X, P);

  // Stop the timer

  double time_b = get_time();
  printf( "Processing time: %16.3f (ms)\n", (time_b-time_a)*1000);

  // Write the data to file

  write_periodogram(filename_out, N_f, df, P);

  // Free up space

  free(P);

  free(X);
  free(t);

  // Finish

}


////
// Initialization
////

void
initialize (int argc, char **argv, char **filename_in, char **filename_out, 
	    float *F_over, float *F_high, int *device)
{

  // Set up the argtable structs

  struct arg_file *in = arg_file1(NULL, "in", "<filename_in>", "input file");
  struct arg_file *out = arg_file1(NULL, "out", "<filename_out>", "output file");
  struct arg_dbl *over = arg_dbl0(NULL, "over", "<F_over>", "oversample factor");
  struct arg_dbl *high = arg_dbl0(NULL, "high", "<F_high>", "highest-frequency factor");
  struct arg_int *dev = arg_int0(NULL, "device", "<device>", "device number");
  struct arg_end *end = arg_end(20);
  void *argtable[] = {in,out,over,high,dev,end};

  // Parse the command line

  int n_error = arg_parse(argc, argv, argtable);

  if (n_error == 0) {

    *filename_in = (char *) malloc(strlen(in->filename[0])+1);
    strcpy(*filename_in, in->filename[0]);

    *filename_out = (char *) malloc(strlen(out->filename[0])+1);
    strcpy(*filename_out, out->filename[0]);

    *F_over = over->count == 1 ? (float) over->dval[0] : 1.f;
    *F_high = high->count == 1 ? (float) high->dval[0] : 1.f;

    *device = dev->count == 1 ? dev->ival[0] : 0;

  }
  else {

    printf("Syntax: %s", argv[0]);
    arg_print_syntax(stdout, argtable, "\n");

    exit(EXIT_FAILURE);

  }

  // Finish

}


////
// CUDA Initialization
////

void
initialize_cuda (int device, int N_f)
{

  // Select the device

  CUDA_CALL(hipSetDevice(device));

  // Dummy call to initialize the CUDA runtime
  
  CUDA_CALL(hipDeviceSynchronize());

  // Finish

}


////
// Periodogram evaluation
////

void
eval_LS_periodogram (int N_t, int N_f, float df, 
		     float *t, float *X, float *P)
{

  // Allocate device memory and copy data over

  float *d_t;
  float *d_X;
  float *d_P;

  CUDA_CALL(hipMalloc((void**) &d_t, N_t*sizeof(float)));
  CUDA_CALL(hipMalloc((void**) &d_X, N_t*sizeof(float)));
  CUDA_CALL(hipMalloc((void**) &d_P, N_f*sizeof(float)));

  CUDA_CALL(hipMemcpy(d_t, t, N_t*sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(d_X, X, N_t*sizeof(float), hipMemcpyHostToDevice));

  // Set up run parameters

  dim3 grid_dim(N_f/BLOCK_SIZE, 1, 1);
  dim3 block_dim(BLOCK_SIZE, 1, 1);

  printf("Grid of %d frequency blocks of size %d threads\n", N_f/BLOCK_SIZE, BLOCK_SIZE);

  // Launch the kernel

  printf("Launching kernel...\n");

  culsp_kernel<<<grid_dim, block_dim>>>(d_t, d_X, d_P, df, N_t);

  hipError_t err = hipGetLastError();
  if(err != hipSuccess) {
    fprintf(stderr, "Cuda error: kernel launch failed in file '%s' in line %i : %s.\n",
	    __FILE__, __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
  }

  CUDA_CALL(hipDeviceSynchronize());

  printf("Completed!\n");

  // Copy data from the device

  CUDA_CALL(hipMemcpy(P, d_P, N_f*sizeof(float), hipMemcpyDeviceToHost));

  CUDA_CALL(hipFree(d_P));
  CUDA_CALL(hipFree(d_X));
  CUDA_CALL(hipFree(d_t));

  // Finish

}
